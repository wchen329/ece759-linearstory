#include "hip/hip_runtime.h"
// #include <__clang_cuda_builtin_vars.h>
#include <cstdio>
#include <iostream>
#include <hip/hip_runtime.h>
#include "jacobi_kernel.cuh"
#include <math.h>
#include <vector>
 
void print_arr(float* arr, int n)
{
    for (int i = 0; i < n; i++) printf("%f ", arr[i]);
    printf("\n");
}

__global__ void jacobi_kernel(float *A, float *B, float *x, float *x_new, int n)
{
	int tx = blockIdx.x*blockDim.x + threadIdx.x;
	if (tx >= n) return;


	extern __shared__ float shared_memory[];
	float *shared_x = shared_memory;
	float *shared_x_new = shared_x + blockDim.x;
	float *shared_B = shared_x_new + blockDim.x;

	shared_x[threadIdx.x] = x[tx];
	shared_B[threadIdx.x] = B[tx];
	__syncthreads();

	float LU_sum = 0;
	
	for (int j = 0; j < n; j++)
	{
		LU_sum += A[tx*n + j]*x[j];
		// LU_sum += A[tx*n + j]*x[j];
	}
	LU_sum -= A[tx*n+tx]*x[threadIdx.x];
	shared_x_new[threadIdx.x] = (shared_B[threadIdx.x] - LU_sum)/A[tx*n + tx];
	// LU_sum -= A[tx*n+tx]*x[tx];
	// x_new[tx] = (B[tx] - LU_sum)/A[tx*n + tx];

	__syncthreads();
	x_new[tx] = shared_x_new[threadIdx.x];


}

void jacobi(float *A, float *B, float *x, int n, int threads_per_block)
{
	const int ITERATION_LIMIT = 200;

	int numBlocks = (int)ceil( (double)n/(double)threads_per_block);
	int shared_space = sizeof(float)*3*threads_per_block; // for x, x_new, and B
	
	float *dA, *dB, *dx, *dx_new;

	hipMalloc((void**)&dA, sizeof(float) * n * n);
	hipMalloc((void**)&dB, sizeof(float) * n);
	hipMalloc((void**)&dx, sizeof(float) * n);
	hipMalloc((void**)&dx_new, sizeof(float) * n);

	hipMemcpy(dA, A, sizeof(float) * n * n, hipMemcpyHostToDevice);
	hipMemcpy(dB, B, sizeof(float) * n, hipMemcpyHostToDevice);
	hipMemcpy(dx, x, sizeof(float) * n, hipMemcpyHostToDevice);

	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipEventRecord(start);

	for (int k = 0; k < ITERATION_LIMIT/2; k++)
    {
		jacobi_kernel<<<numBlocks, threads_per_block, shared_space>>>(dA, dB, dx, dx_new, n);
		jacobi_kernel<<<numBlocks, threads_per_block, shared_space>>>(dA, dB, dx_new, dx, n);

		// hipMemcpy(dx, dx_new, sizeof(float) * n, hipMemcpyDeviceToDevice);
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	float ms;
	hipEventElapsedTime(&ms, start, stop);

	hipMemcpy(x, dx, sizeof(float) * n, hipMemcpyDeviceToHost);

	printf("ms: %f \n", ms);
    // printf("Result: ");
	// print_arr(x, n);

	hipFree(dA);
	hipFree(dB);
	hipFree(dx);
	hipFree(dx_new);

	return;
}

